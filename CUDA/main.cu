#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <time.h>
#include <sys/time.h>
#include <vector>

#define ARRAY_SIZE 100000
#define BLOCK_SIZE 256

int cmpfunc(const void *a, const void *b) {
    return (*(int *)a - *(int *)b);
}


void thrustSortGPU(int *arr, int n) {
    thrust::device_vector<int> d_arr(arr, arr + n);

    thrust::sort(d_arr.begin(), d_arr.end());

    hipMemcpy(arr, thrust::raw_pointer_cast(d_arr.data()), n * sizeof(int), hipMemcpyDeviceToHost);
}


__global__ void mergeSortAndMerge1(int *arr, int n, int curr_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int left_start = tid * 2 * curr_size;
    int mid = left_start + curr_size - 1;
    int right_end = min(left_start + 2 * curr_size - 1, n - 1);

    if (left_start < n - 1) {

        int i, j, k;
        int n1 = mid - left_start + 1;
        int n2 = right_end - mid;

        int *L = &arr[left_start];
        int *R = &arr[mid + 1];

        int *tempL = new int[n1];
        int *tempR = new int[n2];

        for (i = 0; i < n1; i++)
            tempL[i] = L[i];
        for (j = 0; j < n2; j++)
            tempR[j] = R[j];

        i = 0;
        j = 0;
        k = left_start;
        while (i < n1 && j < n2) {
            if (tempL[i] <= tempR[j]) {
                arr[k] = tempL[i];
                i++;
            } else {
                arr[k] = tempR[j];
                j++;
            }
            k++;
        }

        while (i < n1) {
            arr[k] = tempL[i];
            i++;
            k++;
        }

        while (j < n2) {
            arr[k] = tempR[j];
            j++;
            k++;
        }

        delete[] tempL;
        delete[] tempR;
    }
}


void mergeSortAndMergeGPU1(int *arr, int n) {
    int *d_arr;


    hipMalloc((void**)&d_arr, n * sizeof(int));


    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int curr_size = 1;

    while (curr_size <= n - 1) {
        int num_blocks = (n - 1) / (2 * curr_size) + 1;
        mergeSortAndMerge1<<<num_blocks, 1>>>(d_arr, n, curr_size);
        hipDeviceSynchronize(); 
        curr_size = 2 * curr_size;
    }

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_arr);
}





__global__ void mergeSortAndMerge(int *arr, int *temp, int n, int curr_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int left_start = tid * 2 * curr_size;
    int mid = left_start + curr_size - 1;
    int right_end = min(left_start + 2 * curr_size - 1, n - 1);

    if (left_start < n) { 

        int i = left_start;
        int j = mid + 1;
        int k = left_start; 

        while (i <= mid && j <= right_end) {
            if (arr[i] <= arr[j]) {
                temp[k] = arr[i];
                i++;
            } else {
                temp[k] = arr[j];
                j++;
            }
            k++;
        }

        while (i <= mid) {
            temp[k] = arr[i];
            i++;
            k++;
        }

        while (j <= right_end) {
            temp[k] = arr[j];
            j++;
            k++;
        }

        for (i = left_start; i <= right_end; i++) {
            arr[i] = temp[i];
        }
    }
}

void mergeSortAndMergeGPU(int *arr, int n) {
    thrust::device_vector<int> d_arr(arr, arr + n);
    thrust::device_vector<int> d_temp(n);

    int curr_size = 1;
    while (curr_size <= n - 1) {
        int num_blocks = (n - 1) / (2 * curr_size) + 1;
        mergeSortAndMerge<<<num_blocks, BLOCK_SIZE>>>(
            thrust::raw_pointer_cast(d_arr.data()),
            thrust::raw_pointer_cast(d_temp.data()),
            n,
            curr_size);
        hipDeviceSynchronize(); 
        curr_size = 2 * curr_size;
    }

    thrust::copy(d_arr.begin(), d_arr.end(), arr);
}



int isSorted(int *arr, int size) {
    for (int i = 1; i < size; i++) {
        if (arr[i - 1] > arr[i]) {
            return 0; 
        }
    }
    return 1; 
}

int64_t micros() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000 + tv.tv_usec;
}

int main() {
    int arr[ARRAY_SIZE];
    int ref_arr[ARRAY_SIZE];
    int ref_arr2[ARRAY_SIZE];
    int ref_arr3[ARRAY_SIZE];

    srand(time(NULL));
    for (int i = 0; i < ARRAY_SIZE; i++) {
        arr[i] = rand() % 100;
    }
    int n = sizeof(arr) / sizeof(arr[0]);

    memcpy(ref_arr, arr, sizeof(arr));
    memcpy(ref_arr2, arr, sizeof(arr));
    memcpy(ref_arr3, arr, sizeof(arr));
    int64_t start, end;
    double dif;

    printf("\r\nBEGIN TEST\r\n");

    start = micros();
    //mergeSortAndMergeGPU(arr, ARRAY_SIZE);
    thrustSortGPU(arr, ARRAY_SIZE);
    end = micros();
    dif = (double)(end - start) / 1000;
    printf("thrust::sort on GPU completed, %.3lf ms elapsed\n", dif);

    if (isSorted(arr, ARRAY_SIZE)) {
        if (ARRAY_SIZE != sizeof(arr) / sizeof(arr[0])) {
            printf("Array size GPU ERROR\n");
        }
        printf("Sorting on GPU is correct\n");
    } else {
        printf("Sorting on GPU is incorrect\n");
    }

    start = micros();
    qsort(ref_arr, n, sizeof(ref_arr[0]), cmpfunc);
    end = micros();
    dif = (double)(end - start) / 1000;
    printf("Reference qsort completed, %.3lf ms elapsed\n", dif);

    if (isSorted(ref_arr, ARRAY_SIZE)) {
        printf("Reference sorting is correct\n");
    } else {
        printf("Reference sorting is incorrect\n");
    }

    if (memcmp(arr, ref_arr, sizeof(arr)) == 0) {
        printf("\r\nSorting equivalent\r\n");
    } else {
        printf("\r\nSorting ERROR!!!!!!!!\r\n");
    }

    start = micros();
    mergeSortAndMergeGPU1(ref_arr3, ARRAY_SIZE);
    end = micros();
    dif = (double)(end - start) / 1000;
    printf("MergeSort no thrust lib on GPU completed, %.3lf ms elapsed\n", dif);

    if (isSorted(ref_arr3, ARRAY_SIZE)) {
        printf("Reference sorting is correct\n");
    } else {
        printf("Reference sorting is incorrect\n");
    }

    if (memcmp(arr, ref_arr3, sizeof(arr)) == 0) {
        printf("\r\nSorting equivalent\r\n");
    } else {
        printf("\r\nSorting ERROR!!!!!!!!\r\n");
    }




    start = micros();
    mergeSortAndMergeGPU(ref_arr2, ARRAY_SIZE);
    end = micros();
    dif = (double)(end - start) / 1000;
    printf("MergeSort on GPU completed, %.3lf ms elapsed\n", dif);

    if (isSorted(ref_arr2, ARRAY_SIZE)) {
        printf("Reference sorting is correct\n");
    } else {
        printf("Reference sorting is incorrect\n");
    }

    if (memcmp(arr, ref_arr2, sizeof(arr)) == 0) {
        printf("\r\nSorting equivalent\r\n");
    } else {
        printf("\r\nSorting ERROR!!!!!!!!\r\n");
    }


    printf("\r\nEND TEST\r\n");

    return 0;
}
