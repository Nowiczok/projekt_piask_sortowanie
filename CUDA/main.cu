/**
* File: hello_gpu.cu
**/
/* Kernel – does nothing*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// must be power of 2!
#define WORKERS 32 
#define ARRAY_LEN (1024*1024*1024)

__global__ void mykernel(void) {
}
int main(void) {
 mykernel<<<1,1>>>(); /* Launch mykernel on GPU */
 printf("Hello GPU!\n");
 return 0;
}